#include "cuda_helpers.h"
#include "gtest/gtest.h"

TEST(answerToEverythingTest, Right)
{
	EXPECT_EQ(42, cuda_helpers::answerToEverything());
}

TEST(executeKernel, dummy)
{
	dummyKernel <<< 1, 1 >>>();
	EXPECT_EQ(hipSuccess, hipGetLastError());
}

TEST(MemoryCopy, linear)
{
	int h_a = 1;
	int h_b = 1;
	int h_c = 0;

	int *d_a, *d_b, *d_c;

	EXPECT_EQ(hipSuccess, hipMalloc(&d_a, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_b, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_c, sizeof(int)));

	EXPECT_EQ(hipSuccess, hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice));
	EXPECT_EQ(hipSuccess, hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice));

	addInt << <1, 1 >> >(d_a, d_b, d_c);
	EXPECT_EQ(hipSuccess, hipGetLastError());

	EXPECT_EQ(hipSuccess, hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost));
	EXPECT_EQ(2, h_c);

	EXPECT_EQ(hipSuccess, hipFree(d_a));
	EXPECT_EQ(hipSuccess, hipFree(d_b));
	EXPECT_EQ(hipSuccess, hipFree(d_c));
}

int main(int argc, char** argv)
{
	::testing::InitGoogleTest(&argc, argv);
	return RUN_ALL_TESTS();
}
