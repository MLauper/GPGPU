#include "cuda_helpers.h"
#include "gtest/gtest.h"

#define EXPECT_CUDA_SUCCES (val) (EXPECT_EQ(hipSuccess, val))

TEST(answerToEverythingTest, Right)
{
	EXPECT_EQ(42, cuda_helpers::answerToEverything());
}

TEST(executeKernel, dummy)
{
	dummyKernel <<< 1, 1 >>>();
	EXPECT_EQ(hipSuccess, hipGetLastError());
}

TEST(LinearMemory, singleValueCopy)
{
	auto h_a = 1;
	auto h_b = 1;
	auto h_c = 0;

	int *d_a, *d_b, *d_c;

	EXPECT_EQ(hipSuccess, hipMalloc(&d_a, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_b, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_c, sizeof(int)));

	EXPECT_EQ(hipSuccess, hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice));
	EXPECT_EQ(hipSuccess, hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice));

	addInt << <1, 1 >> >(d_a, d_b, d_c);
	EXPECT_EQ(hipSuccess, hipGetLastError());

	EXPECT_EQ(hipSuccess, hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost));
	EXPECT_EQ(2, h_c);

	EXPECT_EQ(hipSuccess, hipFree(d_a));
	EXPECT_EQ(hipSuccess, hipFree(d_b));
	EXPECT_EQ(hipSuccess, hipFree(d_c));
}

TEST(LinearMemory, sharedMemory)
{
	auto h_a = 1;
	auto h_b = 1;
	auto h_c = 0;

	int *d_a, *d_b, *d_c;

	EXPECT_EQ(hipSuccess, hipMalloc(&d_a, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_b, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_c, sizeof(int)));

	EXPECT_EQ(hipSuccess, hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice));
	EXPECT_EQ(hipSuccess, hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice));

	addIntSharedMemory << <1, 1 >> >(d_a, d_b, d_c);
	EXPECT_EQ(hipSuccess, hipGetLastError());

	EXPECT_EQ(hipSuccess, hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost));
	EXPECT_EQ(2, h_c);

	EXPECT_EQ(hipSuccess, hipFree(d_a));
	EXPECT_EQ(hipSuccess, hipFree(d_b));
	EXPECT_EQ(hipSuccess, hipFree(d_c));
}

int main(int argc, char** argv)
{
	::testing::InitGoogleTest(&argc, argv);
	return RUN_ALL_TESTS();
}
