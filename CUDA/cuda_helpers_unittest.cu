#include "cuda_helpers.h"
#include "gtest/gtest.h"

#define EXPECT_CUDA_SUCCES (val) (EXPECT_EQ(hipSuccess, val))

/*! \file cuda_helpers_unittest.cu
 *	\brief Test the proper functionality of CUDA and CUDA Helpers.
 *	
 *	This file contains tests based on the Google Test framework to 
 *	test if CUDA works as expected and if the CUDA Helper functions
 *	properly work.
 */

/*! \brief Test if a CUDA kernel can be scheduled
 *
 *	fefwefew
 */
TEST(executeKernel, dummy)
{
	dummyKernel <<< 1, 1 >>>();
	EXPECT_EQ(hipSuccess, hipGetLastError());
}

/*! \brief Test if a single value can be copied to and from the GPU*/
TEST(LinearMemory, singleValueCopy)
{
	auto h_a = 1;
	auto h_b = 1;
	auto h_c = 0;

	int *d_a, *d_b, *d_c;

	EXPECT_EQ(hipSuccess, hipMalloc(&d_a, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_b, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_c, sizeof(int)));

	EXPECT_EQ(hipSuccess, hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice));
	EXPECT_EQ(hipSuccess, hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice));

	addInt << <1, 1 >> >(d_a, d_b, d_c);
	EXPECT_EQ(hipSuccess, hipGetLastError());

	EXPECT_EQ(hipSuccess, hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost));
	EXPECT_EQ(2, h_c);

	EXPECT_EQ(hipSuccess, hipFree(d_a));
	EXPECT_EQ(hipSuccess, hipFree(d_b));
	EXPECT_EQ(hipSuccess, hipFree(d_c));
}

/*! \brief Test if shared memory can be used.*/
TEST(LinearMemory, sharedMemory)
{
	auto h_a = 1;
	auto h_b = 1;
	auto h_c = 0;

	int *d_a, *d_b, *d_c;

	EXPECT_EQ(hipSuccess, hipMalloc(&d_a, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_b, sizeof(int)));
	EXPECT_EQ(hipSuccess, hipMalloc(&d_c, sizeof(int)));

	EXPECT_EQ(hipSuccess, hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice));
	EXPECT_EQ(hipSuccess, hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice));

	addIntSharedMemory << <1, 1 >> >(d_a, d_b, d_c);
	EXPECT_EQ(hipSuccess, hipGetLastError());

	EXPECT_EQ(hipSuccess, hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost));
	EXPECT_EQ(2, h_c);

	EXPECT_EQ(hipSuccess, hipFree(d_a));
	EXPECT_EQ(hipSuccess, hipFree(d_b));
	EXPECT_EQ(hipSuccess, hipFree(d_c));
}

/*! \brief All tests are run when the Test Executable is run.*/
int main(int argc, char** argv)
{
	::testing::InitGoogleTest(&argc, argv);
	return RUN_ALL_TESTS();
}
