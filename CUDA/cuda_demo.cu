#include "cuda_demo.h"
#include "cuda_helpers.h"
#include <iostream>

int main(int argc, char** argv)
{
	cuda_demo::device_memory::demonstrateDeviceMemory();

	return 0;
}

void cuda_demo::device_memory::demonstrateDeviceMemory()
{
	std::cout << "Device Memory Demo\n";
	std::cout << "==================\n\n";

	linear_memory::demonstrateLinearDeviceMemory();
	linear_memory::demonstrateSharedDeviceMemory();
}

void cuda_demo::device_memory::linear_memory::demonstrateLinearDeviceMemory()
{
	std::cout << "Linear Device Memory Demo\n";
	std::cout << "\tAdding numbers on the GPU:\n";

	// Setting Host Memory Variables
	auto h_a = 1, h_b = 1, h_c = 0;

	// Reserve pointers on Host and allocate memory on device
	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));
	hipMalloc(&d_c, sizeof(int));

	// Move input values to the device
	hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

	// Calculate result on the device
	addInt << <1, 1 >> >(d_a, d_b, d_c);

	// Move output value to the host
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	// Free memory on the device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "\t" << h_a << " + " << h_b << " = " << h_c << "\n";
}

void cuda_demo::device_memory::linear_memory::demonstrateSharedDeviceMemory()
{
	std::cout << "Shared Device Memory Demo\n";
	std::cout << "\tAdding numbers on the GPU via Shared Memory:\n";

	// Setting Host Memory Variables
	auto h_a = 1, h_b = 1, h_c = 0;

	// Reserve pointers on Host and allocate memory on device
	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));
	hipMalloc(&d_c, sizeof(int));

	// Move input values to the device
	hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

	// Calculate result on the device
	addIntSharedMemory << <1, 1 >> >(d_a, d_b, d_c);

	// Move output value to the host
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	// Free memory on the device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "\t" << h_a << " + " << h_b << " = " << h_c << "\n";
}
